#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#define NUM_ELEMS 10000000
#define NUM_THREADS_PER_BLOCK 512

// warp-aggregated atomic increment
__device__ int atomicAggInc(int *counter) {
  cg::coalesced_group active = cg::coalesced_threads();

  int mask = active.ballot(1);
  // select the leader
  int leader = __ffs(mask) - 1;

  // leader does the update
  int res = 0;
  if (active.thread_rank() == leader) {
    res = atomicAdd(counter, __popc(mask));
  }

  // broadcast result
  res = active.shfl(res, leader);

  // each thread computes its own value
  return res + __popc(mask & ((1 << active.thread_rank()) - 1));
}

__global__ void filter_arr(int *dst, int *nres, const int *src, int n) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;

  for (int i = id; i < n; i += gridDim.x * blockDim.x) {
    if (src[i] > 0) dst[atomicAggInc(nres)] = src[i];
  }
}

int main(int argc, char **argv) {
  int *data_to_filter, *filtered_data, nres = 0;
  int *d_data_to_filter, *d_filtered_data, *d_nres;

  data_to_filter = reinterpret_cast<int *>(malloc(sizeof(int) * NUM_ELEMS));

  // Generate input data.
  for (int i = 0; i < NUM_ELEMS; i++) {
    data_to_filter[i] = rand() % 20;
  }

  findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipMalloc(&d_data_to_filter, sizeof(int) * NUM_ELEMS));
  checkCudaErrors(hipMalloc(&d_filtered_data, sizeof(int) * NUM_ELEMS));
  checkCudaErrors(hipMalloc(&d_nres, sizeof(int)));

  checkCudaErrors(hipMemcpy(d_data_to_filter, data_to_filter,
                             sizeof(int) * NUM_ELEMS, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_nres, 0, sizeof(int)));

  dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
  dim3 dimGrid((NUM_ELEMS / NUM_THREADS_PER_BLOCK) + 1, 1, 1);

  filter_arr<<<dimGrid, dimBlock>>>(d_filtered_data, d_nres, d_data_to_filter,
                                    NUM_ELEMS);

  checkCudaErrors(
      hipMemcpy(&nres, d_nres, sizeof(int), hipMemcpyDeviceToHost));

  filtered_data = reinterpret_cast<int *>(malloc(sizeof(int) * nres));

  checkCudaErrors(hipMemcpy(filtered_data, d_filtered_data, sizeof(int) * nres,
                             hipMemcpyDeviceToHost));

  int *host_filtered_data =
      reinterpret_cast<int *>(malloc(sizeof(int) * NUM_ELEMS));

  // Generate host output with host filtering code.
  int host_flt_count = 0;
  for (int i = 0; i < NUM_ELEMS; i++) {
    if (data_to_filter[i] > 0) {
      host_filtered_data[host_flt_count++] = data_to_filter[i];
    }
  }

  printf("\nWarp Aggregated Atomics %s \n",
         host_flt_count == nres ? "PASSED" : "FAILED");

  checkCudaErrors(hipFree(d_data_to_filter));
  checkCudaErrors(hipFree(d_filtered_data));
  checkCudaErrors(hipFree(d_nres));
  free(data_to_filter);
  free(filtered_data);
  free(host_filtered_data);
}
